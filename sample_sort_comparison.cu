#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <math.h>

#define PER_BLOCK 128 // Fixed number of threads per block

__global__ void sample_sort(int *A, int N) {
    __shared__ int loc[PER_BLOCK];
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= N) return; // Avoid out-of-bounds access
    int k = threadIdx.x;
    loc[k] = A[i];
    __syncthreads();

    for (int j = 0; j < PER_BLOCK / 2; j++) {
        if (k % 2 == 0 && k < PER_BLOCK - 1) {
            if (loc[k] > loc[k + 1]) {
                int temp = loc[k];
                loc[k] = loc[k + 1];
                loc[k + 1] = temp;
            }
        }
        __syncthreads();

        if (k % 2 == 1 && k < PER_BLOCK - 1) {
            if (loc[k] > loc[k + 1]) {
                int temp = loc[k];
                loc[k] = loc[k + 1];
                loc[k + 1] = temp;
            }
        }
        __syncthreads();
    }

    A[i] = loc[k];
}

__global__ void merge_chunks(int *A, int N, int chunk_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate the start of the two chunks to merge
    int start1 = 2 * idx * chunk_size;
    int start2 = start1 + chunk_size;

    if (start1 >= N || start2 >= N) return;

    int end1 = min(start2, N);
    int end2 = min(start2 + chunk_size, N);

    // Temporary array to hold the merged result
    int *temp = new int[end2 - start1];
    int i = start1, j = start2, k = 0;

    // Merge the two chunks
    while (i < end1 && j < end2) {
        if (A[i] <= A[j]) {
            temp[k++] = A[i++];
        } else {
            temp[k++] = A[j++];
        }
    }
    while (i < end1) temp[k++] = A[i++];
    while (j < end2) temp[k++] = A[j++];

    // Copy merged result back to the array
    for (i = 0; i < k; i++) {
        A[start1 + i] = temp[i];
    }

    delete[] temp;
}


void merge(int *arr, int l, int m, int r) {
    int i, j, k;
    int n1 = m - l + 1;
    int n2 = r - m;

    int *L = (int *)malloc(n1 * sizeof(int));
    int *R = (int *)malloc(n2 * sizeof(int));

    for (i = 0; i < n1; i++)
        L[i] = arr[l + i];
    for (j = 0; j < n2; j++)
        R[j] = arr[m + 1 + j];

    i = 0;
    j = 0;
    k = l;

    while (i < n1 && j < n2) {
        if (L[i] <= R[j]) {
            arr[k] = L[i];
            i++;
        } else {
            arr[k] = R[j];
            j++;
        }
        k++;
    }

    while (i < n1) {
        arr[k] = L[i];
        i++;
        k++;
    }

    while (j < n2) {
        arr[k] = R[j];
        j++;
        k++;
    }

    free(L);
    free(R);
}

void mergeSort(int *arr, int left, int right) {
    if (left < right) {
        int middle = left + (right - left) / 2;
        mergeSort(arr, left, middle);
        mergeSort(arr, middle + 1, right);
        merge(arr, left, middle, right);
    }
}

void printArray(const char *label, int *arr, int size) {
    printf("%s: ", label);
    for (int i = 0; i < size; i++) {
        printf("%d ", arr[i]);
    }
    printf("\n");
}

void printFullArray(const char *label, int *arr, int size) {
    printf("%s:\n", label);
    for (int i = 0; i < size; i++) {
        printf("%d ", arr[i]);
        if ((i + 1) % 10 == 0) { // Print 10 numbers per line
            printf("\n");
        }
    }
    printf("\n");
}

int main(int argc, char *argv[]) {
    int N = 1024; // Default value for N
    if (argc > 1) {
        N = atoi(argv[1]);
    }

    int num_blocks = (int)ceil((float)N / PER_BLOCK);

    printf("Testing with N = %d\n", N);

    struct timeval start_serial, end_serial, start_cuda, end_cuda;

    int *h_A = (int *)malloc(N * sizeof(int));
    int *m_A = (int *)malloc(N * sizeof(int));

    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        int random = rand() % N + 1;
        h_A[i] = random;
        m_A[i] = random;
    }

    // Print array before sorting
    // printArray("Before Sorting", h_A, (N > 20) ? 20 : N);

    // Serial sorting
    gettimeofday(&start_serial, NULL);
    mergeSort(m_A, 0, N - 1);
    gettimeofday(&end_serial, NULL);

    // Print the array sorted by mergeSort
    // printFullArray("Array Sorted by mergeSort", m_A, N);

    size_t size = N * sizeof(int);
    int *d_A;
    hipMalloc(&d_A, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    dim3 threads_per_block(PER_BLOCK);
    dim3 num_of_blocks(num_blocks);

    // CUDA sorting
    gettimeofday(&start_cuda, NULL);
    sample_sort<<<num_of_blocks, threads_per_block>>>(d_A, N);
    hipDeviceSynchronize();

    int chunk_size = PER_BLOCK; // Start with block-sized chunks

    while (chunk_size < N) {
        int num_chunks = (N + 2 * chunk_size - 1) / (2 * chunk_size);
        merge_chunks<<<num_chunks, threads_per_block>>>(d_A, N, chunk_size);
        hipDeviceSynchronize();
        chunk_size *= 2;
    }

    gettimeofday(&end_cuda, NULL);

    hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);

    // Print the entire sorted array
    //printFullArray("Globally Sorted Array", h_A, N);

    hipFree(d_A);

    printf("\nSerial Sort Time: %ld microseconds\n",
           (end_serial.tv_sec - start_serial.tv_sec) * 1000000 +
               (end_serial.tv_usec - start_serial.tv_usec));

    printf("\nCUDA Sort Time: %ld microseconds\n",
           (end_cuda.tv_sec - start_cuda.tv_sec) * 1000000 +
               (end_cuda.tv_usec - start_cuda.tv_usec));

    printf("\nSpeedup (Serial Sort Time / CUDA Sort Time): %.2f\n",
        (float)((end_serial.tv_sec - start_serial.tv_sec) * 1000000 +
                (end_serial.tv_usec - start_serial.tv_usec)) /
        (float)((end_cuda.tv_sec - start_cuda.tv_sec) * 1000000 +
                (end_cuda.tv_usec - start_cuda.tv_usec)));
            
    free(h_A);
    free(m_A);

    return 0;
}
